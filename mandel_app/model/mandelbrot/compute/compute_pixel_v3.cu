#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>
extern "C" __global__
void mandel_pixel(const complex<double>* c,
                  complex<double>* z,
                  int* iterations,
                  const int start_iter,
                  const int end_iter
                 )
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int k = start_iter;
    const double cx = c[tid].real();
    const double cy = c[tid].imag();
    double x;
    double y;
    double xx;
    double yy;
    double xy;
    /* double x2; */
    bool cont = true;
    if (start_iter == 0)
    {
        x = cx;
        y = cy;
    }
    else
    {
        x = z[tid].real();
        y = z[tid].imag();
    }

    if (k == end_iter)
    {
        cont = false;
    }
    else
    {
        xx = x * x;
        yy = y * y;
        cont = (xx + yy < 4.0);
    }

    while (cont)
    {
        xy = x * y;
        /* x2 = x + x; */
        /* x2 = 2 * x; */
        x = xx - yy + cx;
        /*y = 2.0*xy + cy;*/
        y = __fma_rn(xy, 2.0, cy);

        k++;
        
        if (k == end_iter)
        {
            cont = false;
            z[tid] = complex<double>(x, y);
        }
        else
        {
            xx = x * x;
            yy = y * y;
            /* xy = x * y; */
            /*if (xx > 2.0 || yy > 2.0)    possible further speed up
            {*/
            cont = (xx + yy < 4.0);
            /*}*/
        }
    }
    iterations[tid] = k;
}
