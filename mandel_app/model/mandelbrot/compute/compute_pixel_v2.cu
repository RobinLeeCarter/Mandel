#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>
extern "C" __global__
void mandel_pixel(const complex<double>* c,
                  complex<double>* z,
                  int* iterations,
                  const int start_iter,
                  const int end_iter
                 )
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int k = start_iter;
    const double cx = c[tid].real();
    const double cy = c[tid].imag();
    double x;
    double y;
    double x_p_y;
    double x_m_y;
    /* double xy; */
    double x2;
    bool cont = true;
    if (start_iter == 0)
    {
        x = cx;
        y = cy;
    }
    else
    {
        x = z[tid].real();
        y = z[tid].imag();
    }

    if (k == end_iter)
    {
        cont = false;
    }
    else
    {
        cont = (x*x + y*y < 4.0);
        x_p_y = x + y;
        x_m_y = x - y;
    }

    while (cont)
    {
        /* x2 = x + x; */

        /* x = x*x - y*y + cx; */
        /* y = 2*x*y + cy; */
        x2 = x * 2.0;
        x = __fma_rn(x_m_y, x_p_y, cx);
        y = __fma_rn(x2, y, cy);

        k++;
        
        if (k == end_iter)
        {
            cont = false;
            z[tid] = complex<double>(x, y);
        }
        else
        {
            x_p_y = x + y;
            x_m_y = x - y;
            if (x_p_y > 2 || x_p_y < -2 || x_m_y > 2 || x_m_y < -2)
            {
                /* large and rare, outside diamond, potentially outside circle, do full test */
                cont = (x*x + y*y < 4.0);
            }
        }
    }
    iterations[tid] = k;
}
