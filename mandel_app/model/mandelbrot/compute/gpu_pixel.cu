#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>
extern "C" __global__
void do_pixel(const complex<double>* c,
                  complex<double>* z,
                  int* iterations,
                  const int end_iter
                 )
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int k;
    const double cx = c[tid].real();
    const double cy = c[tid].imag();
    double x;
    double y;
    double xx;
    double yy;
    double x2;
    bool cont = true;

    x = z[tid].real();
    y = z[tid].imag();
    k = iterations[tid];

    xx = x * x;
    yy = y * y;
    cont = (k < end_iter && xx + yy < 4.0);

    while (cont)
    {
        // y = 2*x*y + cy;
        x2 = x * 2.0;
        y = __fma_rn(x2, y, cy);
        x = xx - yy + cx;
        k++;

        if (k == end_iter)
        {
            cont = false;
        }
        else
        {
            xx = x * x;
            yy = y * y;
            cont = (xx + yy < 4.0);
        }
    }
    z[tid] = complex<double>(x, y);
    iterations[tid] = k;
}
