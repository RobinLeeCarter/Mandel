#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>
extern "C" __global__
void mandel_pixel(const complex<double>* c,
                  complex<double>* z,
                  int* iterations,
                  const int start_iter,
                  const int end_iter
                 )
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int k = start_iter;
    const double cx = c[tid].real();
    const double cy = c[tid].imag();
    double x;
    double y;
    double xx;
    double yy;
    double x2;
    bool cont = true;

    x = z[tid].real();
    y = z[tid].imag();

    xx = x * x;
    yy = y * y;
    cont = (xx + yy < 4.0);

    while (cont)
    {
        // y = 2*x*y + cy;
        x2 = x * 2.0;
        y = __fma_rn(x2, y, cy);
        x = xx - yy + cx;
        k++;

        if (k == end_iter)
        {
            cont = false;
            z[tid] = complex<double>(x, y);
        }
        else
        {
            xx = x * x;
            yy = y * y;
            cont = (xx + yy < 4.0);
        }
    }
    iterations[tid] = k;
}
