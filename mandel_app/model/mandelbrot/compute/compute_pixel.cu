#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>
extern "C" __global__
void mandel_pixel(const complex<double>* c,
                  complex<double>* z,
                  int* iterations,
                  const int start_iter,
                  const int end_iter
                 )
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int k = start_iter;
    const double cx = c[tid].real();
    const double cy = c[tid].imag();
    double x;
    double y;
    double xx;
    double yy;
    /* double xy; */
    double x2;
    bool cont = true;
    if (start_iter == 0)
    {
        x = cx;
        y = cy;
    }
    else
    {
        x = z[tid].real();
        y = z[tid].imag();
    }

    if (k == end_iter)
    {
        cont = false;
    }
    else
    {
        xx = x * x;
        yy = y * y;
        /* xy = x * y; */
        cont = (xx + yy < 4.0);
    }

    while (cont)
    {
        /* x2 = x + x; */
        x2 = x * 2.0;
        x = xx - yy + cx;
        /* y = 2*x*y + cy; */
        y = __fma_rn(x2, y, cy);

        k++;
        
        if (k == end_iter)
        {
            cont = false;
            z[tid] = complex<double>(x, y);
        }
        else
        {
            xx = x * x;
            yy = y * y;
            /* xy = x * y; */
            cont = (xx + yy < 4.0);
        }
    }
    iterations[tid] = k;
}
